#include <stdio.h>

int main()
{
    hipDeviceProp_t  prop;

    int count;
    hipGetDeviceCount( &count );

    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
		printf( "\n" );

        printf( "   --- SM Information for device %d ---\n", i );
        printf( "SM count:  %d\n", prop.multiProcessorCount );
        printf( "Maximum resident threads per SM:  %d\n", prop.maxThreadsPerMultiProcessor );
        printf( "Maximum number of resident blocks per SM:  %d\n", prop.maxBlocksPerMultiProcessor );
        printf( "Shared mem per SM:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per SM:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n", prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] );
        printf( "\n" );
    }

    return 0;
}